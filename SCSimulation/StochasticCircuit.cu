#include "hip/hip_runtime.h"
﻿#include "cuda_base.cuh"

#include <vector>
#include <algorithm>

#include "StochasticCircuit.cuh"
#include "CircuitComponent.cuh"
#include "StochasticNumber.cuh"

namespace scsim {

	StochasticCircuit::StochasticCircuit(uint32_t sim_length, uint32_t num_nets, uint32_t* net_values, uint32_t* net_progress, uint32_t num_components_comb, uint32_t num_components_seq,
		CircuitComponent** components) :
		host_only(true), sim_length(sim_length), sim_length_words((sim_length + 31) / 32), num_nets(num_nets), net_values_host(net_values), net_values_host_pitch((sim_length + 31) / 32 * sizeof(uint32_t)),
		net_values_dev(nullptr), net_values_dev_pitch(0), net_progress_host(net_progress), net_progress_dev(nullptr), num_components_comb(num_components_comb), num_components_seq(num_components_seq),
		num_components(num_components_comb + num_components_seq), components_host(components), components_dev(nullptr), component_array_host(nullptr), component_array_host_pitch(0),
		component_array_dev(nullptr), component_array_dev_pitch(0) {

	}

	StochasticCircuit::StochasticCircuit(uint32_t sim_length, uint32_t num_nets, uint32_t* net_values_host, uint32_t* net_values_dev, size_t net_values_dev_pitch, uint32_t* net_progress_host, uint32_t* net_progress_dev,
		uint32_t num_components_comb, uint32_t num_components_seq, CircuitComponent** components_host, CircuitComponent** components_dev, char* component_array_host, size_t component_array_host_pitch,
		char* component_array_dev, size_t component_array_dev_pitch) :
		host_only(false), sim_length(sim_length), sim_length_words((sim_length + 31) / 32), num_nets(num_nets), net_values_host(net_values_host),
		net_values_host_pitch((sim_length + 31) / 32 * sizeof(uint32_t)), net_values_dev(net_values_dev), net_values_dev_pitch(net_values_dev_pitch), net_progress_host(net_progress_host),
		net_progress_dev(net_progress_dev), num_components_comb(num_components_comb), num_components_seq(num_components_seq), num_components(num_components_comb + num_components_seq),
		components_host(components_host), components_dev(components_dev), component_array_host(component_array_host), component_array_host_pitch(component_array_host_pitch),
		component_array_dev(component_array_dev), component_array_dev_pitch(component_array_dev_pitch) {

	}

	StochasticCircuit::~StochasticCircuit() {
		if (host_only) {
			for (uint32_t i = 0; i < num_components; i++) {
				delete components_host[i]; //host-only: components are in originally allocated positions, delete
			}
		}
		else {
			for (uint32_t i = 0; i < num_components; i++) {
				components_host[i]->~CircuitComponent(); //device-accelerated: components are in component array, only deconstruct
			}
		}

		free(net_values_host);
		free(net_progress_host);
		free(components_host);

		if (!host_only) {
			hipFree(net_values_dev);
			hipFree(net_progress_dev);
			hipFree(components_dev);
			free(component_array_host);
			hipFree(component_array_dev);
		}
	}

	void StochasticCircuit::reset_circuit() {
		simulation_finished = false;
		memset(net_progress_host, 0, num_nets * sizeof(uint32_t)); //clear progress on all nets

		for (uint32_t i = 0; i < num_components_seq; i++) {
			components_host[num_components_comb + i]->reset_state(); //reset sequential component states
		}
	}

	void StochasticCircuit::set_net_value(uint32_t net, StochasticNumber* value) {
		memcpy((net_values_host + (sim_length_words * net)), value->get_data(), __min(sim_length_words, value->word_length) * sizeof(uint32_t)); //copy data from SN
		net_progress_host[net] = __min(sim_length, value->length);
	}

	void StochasticCircuit::set_net_value_unipolar(uint32_t net, double value, uint32_t length) {
		auto tempnum = StochasticNumber::generate_unipolar(length, value);
		set_net_value(net, tempnum);
		delete tempnum;
	}

	void StochasticCircuit::set_net_value_unipolar(uint32_t net, double value) {
		set_net_value_unipolar(net, value, sim_length);
	}

	void StochasticCircuit::set_net_value_bipolar(uint32_t net, double value, uint32_t length) {
		auto tempnum = StochasticNumber::generate_bipolar(length, value);
		set_net_value(net, tempnum);
		delete tempnum;
	}

	void StochasticCircuit::set_net_value_bipolar(uint32_t net, double value) {
		set_net_value_bipolar(net, value, sim_length);
	}

	void StochasticCircuit::set_net_value_constant(uint32_t net, bool value, uint32_t length) {
		auto actual_length = __min(length, sim_length);
		auto net_value = net_values_host + (sim_length_words * net);

		auto word_length = length / 32;
		if (word_length > 0) memset(net_value, value ? 0xff : 0x00, word_length * sizeof(uint32_t));

		auto extra_length = length % 32;
		if (extra_length > 0) {
			if (value) net_value[word_length] |= (0xffffffff << (32 - extra_length));
			else net_value[word_length] &= 0xffffffff >> extra_length;
		}
	}

	void StochasticCircuit::set_net_value_constant(uint32_t net, bool value) {
		set_net_value_constant(net, value, sim_length);
	}

	void StochasticCircuit::copy_data_to_device() {
		if (host_only) return;

		cu(hipMemcpy2D(net_values_dev, net_values_dev_pitch, net_values_host, net_values_host_pitch, net_values_host_pitch, num_nets, hipMemcpyHostToDevice)); //copy net values
		cu(hipMemcpy(net_progress_dev, net_progress_host, num_nets * sizeof(uint32_t), hipMemcpyHostToDevice)); //copy net progress
		cu(hipMemcpy2D(component_array_dev, component_array_dev_pitch, component_array_host, component_array_host_pitch, component_array_host_pitch, num_components, hipMemcpyHostToDevice)); //copy component array

		for (uint32_t i = 0; i < num_components; i++) components_host[i]->copy_state_host_to_device(); //copy component extended state where required
	}

	void StochasticCircuit::copy_data_from_device() {
		if (host_only) return;

		cu(hipMemcpy2D(net_values_host, net_values_host_pitch, net_values_dev, net_values_dev_pitch, net_values_host_pitch, num_nets, hipMemcpyDeviceToHost)); //copy net values
		cu(hipMemcpy(net_progress_host, net_progress_dev, num_nets * sizeof(uint32_t), hipMemcpyDeviceToHost)); //copy net progress
		cu(hipMemcpy2D(component_array_host, component_array_host_pitch, component_array_dev, component_array_dev_pitch, component_array_host_pitch, num_components, hipMemcpyDeviceToHost)); //copy component array

		for (uint32_t i = 0; i < num_components; i++) components_host[i]->copy_state_device_to_host(); //copy component extended state where required
	}

	void StochasticCircuit::simulate_circuit_host_only() {
		std::vector<uint32_t> last_round_possible_progress(num_components, 0);

		while (!simulation_finished) { //iterate over all components until simulation is finished
			int finished_components = 0;

			for (uint32_t i = 0; i < num_components; i++) { //sequentially go through components
				CircuitComponent* comp = components_host[i];

				comp->calculate_simulation_progress();

				if (comp->next_sim_progress() == last_round_possible_progress[i]) { //check and mark finished components
					finished_components++;
					continue;
				}
				last_round_possible_progress[i] = comp->next_sim_progress();

				comp->simulate_step_host(); //simulate next step
				comp->sim_step_finished();
			}

			if (finished_components == num_components) simulation_finished = true; //done if all components finished
		}
	}

	__global__ void exec_sim_step(CircuitComponent** components, uint32_t* comp_indices, uint32_t* comp_counts, uint32_t* comp_offsets) {
		auto type = blockIdx.z;
		auto comp = blockIdx.x * blockDim.x + threadIdx.x;
		if (comp < comp_counts[type]) components[comp_indices[comp_offsets[type] + comp]]->simulate_step_dev();
	}

	void StochasticCircuit::simulate_circuit() {
		if (host_only) {
			simulate_circuit_host_only();
			return;
		}

		std::vector<uint32_t> last_round_possible_progress(num_components, 0);

		while (!simulation_finished) { //iterate over all components until simulation is finished
			int finished_components = 0;

			//combinatorial components
			std::vector<uint32_t> sim_comb;
			std::vector<uint32_t> comb_type_counts;
			std::vector<uint32_t> comb_type_offsets({ 0 });
			uint32_t last_type = 0;
			uint32_t comb_sim_words = 0;
			for (uint32_t i = 0; i < num_components_comb; i++) {
				CircuitComponent* comp = components_host[i];
				uint32_t ctype = comp->component_type;

				comp->calculate_simulation_progress();

				if (comp->next_sim_progress() == last_round_possible_progress[i]) { //check and mark finished components
					finished_components++;
					continue;
				}
				last_round_possible_progress[i] = comp->next_sim_progress();

				auto words = comp->next_sim_progress_word() - comp->current_sim_progress_word();
				if (words > comb_sim_words) comb_sim_words = words; //remember largest number of words to be simulated

				//arrange components by types and remember counts and offsets
				if (ctype == last_type) {
					comb_type_counts.back()++;
				}
				else {
					if (!comb_type_counts.empty()) comb_type_offsets.push_back(comb_type_offsets.back() + comb_type_counts.back());
					comb_type_counts.push_back(1);
					last_type = ctype;
				}
				sim_comb.push_back(i);
			}

			if (!sim_comb.empty()) { //if combinatorial components need to be simulated
				uint32_t* sim_comb_dev;
				uint32_t* comb_type_counts_dev;
				uint32_t* comb_type_offsets_dev;

				//copy component pointers, counts, offsets to device
				cu(hipMalloc(&sim_comb_dev, sim_comb.size() * sizeof(uint32_t)));
				cu(hipMalloc(&comb_type_counts_dev, comb_type_counts.size() * sizeof(uint32_t)));
				cu(hipMalloc(&comb_type_offsets_dev, comb_type_offsets.size() * sizeof(uint32_t)));
				cu(hipMemcpy(sim_comb_dev, sim_comb.data(), sim_comb.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
				cu(hipMemcpy(comb_type_counts_dev, comb_type_counts.data(), comb_type_counts.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
				cu(hipMemcpy(comb_type_offsets_dev, comb_type_offsets.data(), comb_type_offsets.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

				uint32_t block_size_y = __min(comb_sim_words, 64); //block size y: words to simulate

				auto num_threads_comp = *std::max_element(comb_type_counts.begin(), comb_type_counts.end());
				uint32_t block_size_x = __min(num_threads_comp, 256 / block_size_y); //block size x: components of same type to simulate

				dim3 block_size(block_size_x, block_size_y);

				//grid size x and y: component/word count split into multiple blocks
				//grid size z: component types
				dim3 grid_size((num_threads_comp + block_size_x - 1) / block_size_x, (comb_sim_words + block_size_y - 1) / block_size_y, comb_type_counts.size());

				//transfer data to device, simulate, return data
				copy_data_to_device();
				exec_sim_step<<<grid_size, block_size>>>(components_dev, sim_comb_dev, comb_type_counts_dev, comb_type_offsets_dev);
				copy_data_from_device();

				cu(hipFree(sim_comb_dev));
				cu(hipFree(comb_type_counts_dev));
				cu(hipFree(comb_type_offsets_dev));

				for (auto id : sim_comb) {
					components_host[id]->sim_step_finished(); //mark step as finished
				}
			}

			//sequential components, similar to combinatorial as shown above
			std::vector<uint32_t> sim_seq;
			std::vector<uint32_t> seq_type_counts;
			std::vector<uint32_t> seq_type_offsets({ 0 });
			for (uint32_t i = num_components_comb; i < num_components; i++) {
				CircuitComponent* comp = components_host[i];
				uint32_t ctype = comp->component_type;

				comp->calculate_simulation_progress();

				if (comp->next_sim_progress() == last_round_possible_progress[i]) { //check and mark finished components
					finished_components++;
					continue;
				}
				last_round_possible_progress[i] = comp->next_sim_progress();

				if (ctype == last_type) {
					seq_type_counts.back()++;
				}
				else {
					if (!seq_type_counts.empty()) seq_type_offsets.push_back(seq_type_offsets.back() + seq_type_counts.back());
					seq_type_counts.push_back(1);
					last_type = ctype;
				}
				sim_seq.push_back(i);
			}

			if (!sim_seq.empty()) {
				uint32_t* sim_seq_dev;
				uint32_t* seq_type_counts_dev;
				uint32_t* seq_type_offsets_dev;
				cu(hipMalloc(&sim_seq_dev, sim_seq.size() * sizeof(uint32_t)));
				cu(hipMalloc(&seq_type_counts_dev, seq_type_counts.size() * sizeof(uint32_t)));
				cu(hipMalloc(&seq_type_offsets_dev, seq_type_offsets.size() * sizeof(uint32_t)));
				cu(hipMemcpy(sim_seq_dev, sim_seq.data(), sim_seq.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
				cu(hipMemcpy(seq_type_counts_dev, seq_type_counts.data(), seq_type_counts.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
				cu(hipMemcpy(seq_type_offsets_dev, seq_type_offsets.data(), seq_type_offsets.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

				auto max_num_threads = *std::max_element(seq_type_counts.begin(), seq_type_counts.end());
				uint32_t block_size = __min(max_num_threads, 256); //only one thread per component -> block size y = 1, grid size y = 1
				dim3 grid_size((max_num_threads + block_size - 1) / block_size, 1, seq_type_counts.size());

				copy_data_to_device();
				exec_sim_step<<<grid_size, block_size>>>(components_dev, sim_seq_dev, seq_type_counts_dev, seq_type_offsets_dev);
				copy_data_from_device();

				cu(hipFree(sim_seq_dev));
				cu(hipFree(seq_type_counts_dev));
				cu(hipFree(seq_type_offsets_dev));

				for (auto id : sim_seq) {
					components_host[id]->sim_step_finished();
				}
			}

			if (finished_components == num_components) simulation_finished = true; //done if all components finished
		}
	}

	StochasticNumber* StochasticCircuit::get_net_value(uint32_t net) {
		auto progress = net_progress_host[net];
		if (progress == 0) throw;
		return new StochasticNumber(progress, net_values_host + (sim_length_words * net));
	}

	double StochasticCircuit::get_net_value_unipolar(uint32_t net) {
		auto num = get_net_value(net);
		auto ret = num->get_value_unipolar();
		delete num;
		return ret;
	}

	double StochasticCircuit::get_net_value_bipolar(uint32_t net) {
		auto num = get_net_value(net);
		auto ret = num->get_value_bipolar();
		delete num;
		return ret;
	}

}
