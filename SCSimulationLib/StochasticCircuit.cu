#include "hip/hip_runtime.h"
﻿#include "cuda_base.cuh"

#include <vector>
#include <algorithm>

#include "StochasticCircuit.cuh"
#include "CircuitComponent.cuh"
#include "StochasticNumber.cuh"
#include "Scheduler.h"

namespace scsim {

	StochasticCircuit::StochasticCircuit(uint32_t sim_length, uint32_t num_nets, uint32_t* net_values, uint32_t* net_progress, uint32_t num_components_comb, uint32_t num_components_seq, uint32_t* components_index,
		CircuitComponent** components, uint32_t* component_progress, uint32_t num_component_types, uint32_t* component_io, size_t* component_io_offsets, StochasticNumber* net_numbers) :
		host_only(true), sim_length(sim_length), sim_length_words((sim_length + 31) / 32), num_nets(num_nets), net_values_host(net_values), net_values_host_pitch((sim_length + 31) / 32 * sizeof(uint32_t)),
		net_values_dev(nullptr), net_values_dev_pitch(0), net_progress_host(net_progress), net_progress_dev(nullptr), num_components_comb(num_components_comb), num_components_seq(num_components_seq),
		num_components(num_components_comb + num_components_seq), components_host_index(components_index), components_host(components), components_dev(nullptr), component_array_host(nullptr), component_array_host_pitch(0),
		component_array_dev(nullptr), component_array_dev_pitch(0), component_progress_host(component_progress), component_progress_dev(nullptr), num_component_types(num_component_types),
		component_io_host(component_io), component_io_dev(nullptr), component_io_offsets_host(component_io_offsets), component_io_offsets_dev(nullptr), net_numbers(net_numbers) {

		scheduler = nullptr;
	}

	StochasticCircuit::StochasticCircuit(uint32_t sim_length, uint32_t num_nets, uint32_t* net_values_host, uint32_t* net_values_dev, size_t net_values_dev_pitch, uint32_t* net_progress_host, uint32_t* net_progress_dev,
		uint32_t num_components_comb, uint32_t num_components_seq, uint32_t* components_host_index, CircuitComponent** components_host, CircuitComponent** components_dev, char* component_array_host, size_t component_array_host_pitch,
		char* component_array_dev, size_t component_array_dev_pitch, uint32_t* component_progress_host, uint32_t* component_progress_dev, uint32_t num_component_types, uint32_t* component_io_host,
		uint32_t* component_io_dev, size_t* component_io_offsets_host, size_t* component_io_offsets_dev, StochasticNumber* net_numbers) :
		host_only(false), sim_length(sim_length), sim_length_words((sim_length + 31) / 32), num_nets(num_nets), net_values_host(net_values_host),
		net_values_host_pitch((sim_length + 31) / 32 * sizeof(uint32_t)), net_values_dev(net_values_dev), net_values_dev_pitch(net_values_dev_pitch), net_progress_host(net_progress_host),
		net_progress_dev(net_progress_dev), num_components_comb(num_components_comb), num_components_seq(num_components_seq), num_components(num_components_comb + num_components_seq),
		components_host_index(components_host_index), components_host(components_host), components_dev(components_dev), component_array_host(component_array_host), component_array_host_pitch(component_array_host_pitch),
		component_array_dev(component_array_dev), component_array_dev_pitch(component_array_dev_pitch), component_progress_host(component_progress_host),
		component_progress_dev(component_progress_dev), num_component_types(num_component_types), component_io_host(component_io_host), component_io_dev(component_io_dev),
		component_io_offsets_host(component_io_offsets_host), component_io_offsets_dev(component_io_offsets_dev), net_numbers(net_numbers) {

		scheduler = nullptr;
	}

	StochasticCircuit::~StochasticCircuit() {
		if (host_only) {
			for (uint32_t i = 0; i < num_components; i++) {
				delete components_host[i]; //host-only: components are in originally allocated positions, delete
			}
		}
		else {
			for (uint32_t i = 0; i < num_components; i++) {
				components_host[i]->~CircuitComponent(); //device-accelerated: components are in component array, only deconstruct
			}
		}

		for (uint32_t i = 0; i < num_nets; i++) {
			net_numbers[i].~StochasticNumber();
		}

		if (!host_only) {
			cu_ignore_error(hipFree(net_values_dev));
			cu_ignore_error(hipFree(net_progress_dev));
			cu_ignore_error(hipFree(components_dev));
			cu_ignore_error(hipFree(component_array_dev));
			cu_ignore_error(hipFree(component_progress_dev));
			cu_ignore_error(hipFree(component_io_dev));
			cu_ignore_error(hipFree(component_io_offsets_dev));

			cu_ignore_error(hipHostUnregister(component_array_host));
			cu_ignore_error(hipHostUnregister(net_values_host));
			cu_ignore_error(hipHostUnregister(net_progress_host));
			cu_ignore_error(hipHostUnregister(component_progress_host));

			free(component_array_host);
		}

		free(net_values_host);
		free(net_progress_host);
		free(components_host);
		free(component_progress_host);
		free(component_io_host);
		free(component_io_offsets_host);
		free(net_numbers);

		delete scheduler;
	}

	void StochasticCircuit::reset_circuit() {
		simulation_finished = false;
		memset(net_progress_host, 0, num_nets * sizeof(uint32_t)); //clear progress on all nets
		memset(component_progress_host, 0, 2 * num_components * sizeof(uint32_t)); //clear progress on all components
		if (!host_only) {
			cu(hipMemset(net_progress_dev, 0, num_nets * sizeof(uint32_t)));
			cu(hipMemset(component_progress_dev, 0, 2 * num_components * sizeof(uint32_t)));
		}

		for (uint32_t i = 0; i < num_components_seq; i++) {
			components_host[num_components_comb + i]->reset_state(); //reset sequential component states
		}
	}

	void StochasticCircuit::set_net_value(uint32_t net, StochasticNumber& value) {
		if (net >= num_nets) throw std::runtime_error("set_net_value: Invalid net index.");
		if (value.length() > sim_length) throw std::runtime_error("set_net_value: SN length exceeds the simulation time span.");

		net_numbers[net] = value;
	}

	void StochasticCircuit::set_net_value_unipolar(uint32_t net, double value, uint32_t length) {
		if (net >= num_nets) throw std::runtime_error("set_net_value_unipolar: Invalid net index.");
		if (length > sim_length) throw std::runtime_error("set_net_value_unipolar: Length exceeds the simulation time span.");

		net_numbers[net].set_length(length);
		net_numbers[net].set_value_unipolar(value);
	}

	void StochasticCircuit::set_net_value_unipolar(uint32_t net, double value) {
		set_net_value_unipolar(net, value, sim_length);
	}

	void StochasticCircuit::set_net_value_bipolar(uint32_t net, double value, uint32_t length) {
		if (net >= num_nets) throw std::runtime_error("set_net_value_bipolar: Invalid net index.");
		if (length > sim_length) throw std::runtime_error("set_net_value_bipolar: Length exceeds the simulation time span.");

		net_numbers[net].set_length(length);
		net_numbers[net].set_value_bipolar(value);
	}

	void StochasticCircuit::set_net_value_bipolar(uint32_t net, double value) {
		set_net_value_bipolar(net, value, sim_length);
	}

	void StochasticCircuit::set_net_values_curand(uint32_t* nets, double* values_unipolar, uint32_t count, uint32_t length, bool copy) {
		if (host_only) throw std::runtime_error("set_net_values_curand: Cannot be used in host-only circuits.");
		if (length == 0) throw std::runtime_error("set_net_values_curand: Length must be greater than zero.");
		if (length > sim_length) throw std::runtime_error("set_net_values_curand: Length exceeds the simulation time span.");
		if (count == 0) throw std::runtime_error("set_net_values_curand: Count must be greater than zero.");

		auto netvalue_ptrs = (uint32_t**)malloc(count * sizeof(uint32_t*)); //calculate device pointers for net data
		for (uint32_t i = 0; i < count; i++) {
			uint32_t net = nets[i];
			if (net >= num_nets) throw std::runtime_error("set_net_values_curand: Invalid net index.");
			netvalue_ptrs[i] = (uint32_t*)((char*)net_values_dev + (net * net_values_dev_pitch));
		}

		auto word_length = (length + 31) / 32;

		uint32_t max_batch = MAX_CURAND_BATCH_WORDS / word_length; //number of nets in one batch

		if (copy) copy_data_to_device(length);

		for (uint32_t batch_offset = 0; batch_offset < count; batch_offset += max_batch) { //generate net values
			uint32_t batch_size = __min(count - batch_offset, max_batch);

			StochasticNumber::generate_bitstreams_curand(netvalue_ptrs + batch_offset, length, values_unipolar + batch_offset, batch_size);
		}

		if (copy) copy_data_from_device(length);

		for (uint32_t i = 0; i < count; i++) { //update net progress
			net_progress_host[nets[i]] = length;
		}

		free(netvalue_ptrs);
	}

	void StochasticCircuit::set_net_values_curand(uint32_t* nets, double* values_unipolar, uint32_t count, bool copy) {
		set_net_values_curand(nets, values_unipolar, count, sim_length, copy);
	}

	void StochasticCircuit::set_net_values_curand(uint32_t first_net, double* values_unipolar, uint32_t count, uint32_t length, bool copy) {
		if (host_only) throw std::runtime_error("set_net_values_curand: Cannot be used in host-only circuits.");
		if (length == 0) throw std::runtime_error("set_net_values_curand: Length must be greater than zero.");
		if (length > sim_length) throw std::runtime_error("set_net_values_curand: Length exceeds the simulation time span.");
		if (count == 0) throw std::runtime_error("set_net_values_curand: Count must be greater than zero.");
		if (first_net + count > num_nets) throw std::runtime_error("set_net_values_curand: Invalid first net index and/or too many nets.");

		auto netvalue_ptrs = (uint32_t**)malloc(count * sizeof(uint32_t*)); //calculate device pointers for net data
		for (uint32_t i = 0; i < count; i++) {
			netvalue_ptrs[i] = (uint32_t*)((char*)net_values_dev + ((first_net + i) * net_values_dev_pitch));
		}

		auto word_length = (length + 31) / 32;

		uint32_t max_batch = MAX_CURAND_BATCH_WORDS / word_length; //number of nets in one batch

		if (copy) copy_data_to_device(length);

		for (uint32_t batch_offset = 0; batch_offset < count; batch_offset += max_batch) { //generate net values
			uint32_t batch_size = __min(count - batch_offset, max_batch);

			StochasticNumber::generate_bitstreams_curand(netvalue_ptrs + batch_offset, length, values_unipolar + batch_offset, batch_size);
		}

		if (copy) copy_data_from_device(length);

		for (uint32_t i = 0; i < count; i++) { //update net progress
			net_progress_host[first_net + i] = length;
		}

		free(netvalue_ptrs);
	}

	void StochasticCircuit::set_net_values_curand(uint32_t first_net, double* values_unipolar, uint32_t count, bool copy) {
		set_net_values_curand(first_net, values_unipolar, count, sim_length, copy);
	}

	void StochasticCircuit::set_net_value_constant(uint32_t net, bool value, uint32_t length) {
		if (net >= num_nets) throw std::runtime_error("set_net_value_constant: Invalid net index.");
		if (length > sim_length) throw std::runtime_error("set_net_value_constant: Length exceeds the simulation time span.");

		net_numbers[net].set_length(length);
		net_numbers[net].set_value_constant(value);
	}

	void StochasticCircuit::set_net_value_constant(uint32_t net, bool value) {
		set_net_value_constant(net, value, sim_length);
	}

	void StochasticCircuit::copy_data_to_device() {
		copy_data_to_device(sim_length);
	}

	void StochasticCircuit::copy_data_to_device(uint32_t net_length) {
		if (host_only) return;

		size_t width = (__min(net_length, sim_length) + 31) / 32 * sizeof(uint32_t);

		cu(hipMemcpy2D(net_values_dev, net_values_dev_pitch, net_values_host, net_values_host_pitch, width, num_nets, hipMemcpyHostToDevice)); //copy net values
		cu(hipMemcpy(net_progress_dev, net_progress_host, num_nets * sizeof(uint32_t), hipMemcpyHostToDevice)); //copy net progress
		cu(hipMemcpy2D(component_array_dev, component_array_dev_pitch, component_array_host, component_array_host_pitch, component_array_host_pitch, num_components, hipMemcpyHostToDevice)); //copy component array
		cu(hipMemcpy(component_progress_dev, component_progress_host, 2 * num_components * sizeof(uint32_t), hipMemcpyHostToDevice));
	}

	void StochasticCircuit::copy_data_from_device() {
		copy_data_from_device(sim_length);
	}

	void StochasticCircuit::copy_data_from_device(uint32_t net_length) {
		if (host_only) return;

		size_t width = (__min(net_length, sim_length) + 31) / 32 * sizeof(uint32_t);

		cu(hipMemcpy2D(net_values_host, net_values_host_pitch, net_values_dev, net_values_dev_pitch, width, num_nets, hipMemcpyDeviceToHost)); //copy net values
		cu(hipMemcpy(net_progress_host, net_progress_dev, num_nets * sizeof(uint32_t), hipMemcpyDeviceToHost)); //copy net progress
		cu(hipMemcpy2D(component_array_host, component_array_host_pitch, component_array_dev, component_array_dev_pitch, component_array_host_pitch, num_components, hipMemcpyDeviceToHost)); //copy component array
		copy_component_progress_from_device();
	}

	void StochasticCircuit::simulate_circuit_host_only() {
		if (scheduler != nullptr && scheduler->execute(true)) return; //use scheduler if applicable

		std::vector<uint32_t> last_round_possible_progress(num_components, 0);

		while (!simulation_finished) { //iterate over all components until simulation is finished
			int finished_components = 0;

			for (uint32_t i = 0; i < num_components; i++) { //sequentially go through components
				CircuitComponent* comp = components_host[i];

				comp->calculate_simulation_progress_host();

				if (comp->next_sim_progress() == last_round_possible_progress[i]) { //check and mark finished components
					finished_components++;
					continue;
				}
				last_round_possible_progress[i] = comp->next_sim_progress();

				comp->simulate_step_host(); //simulate next step
				comp->sim_step_finished_host();
			}

			if (finished_components == num_components) simulation_finished = true; //done if all components finished
		}
	}

	void StochasticCircuit::simulate_circuit() {
		if (host_only) {
			simulate_circuit_host_only();
			return;
		}

		copy_data_to_device();
		simulate_circuit_dev_nocopy();
		copy_data_from_device();
	}

	__global__ void calc_sim_progress(CircuitComponent** components, uint32_t count) {
		auto comp = blockIdx.x * blockDim.x + threadIdx.x;
		if (comp < count) components[comp]->calculate_simulation_progress_dev();
	}

	__global__ void exec_sim_step(CircuitComponent** components, uint32_t* comp_indices, uint32_t* comp_counts, uint32_t* comp_offsets) {
		auto type = blockIdx.z;
		auto comp = blockIdx.x * blockDim.x + threadIdx.x;
		if (comp < comp_counts[type]) components[comp_indices[comp_offsets[type] + comp]]->simulate_step_dev();
	}

	__global__ void finish_sim_step(CircuitComponent** components, uint32_t* comp_indices, uint32_t count) {
		auto comp = blockIdx.x * blockDim.x + threadIdx.x;
		if (comp < count) components[comp_indices[comp]]->sim_step_finished_dev();
	}

	void StochasticCircuit::simulate_circuit_dev_nocopy() {
		if (host_only) throw std::runtime_error("simulate_circuit_dev_nocopy: This function is not supported for host-only circuits.");

		if (scheduler != nullptr && scheduler->execute(false)) return; //use scheduler if applicable

		uint32_t block_size_calcp = __min(num_components, 256);
		uint32_t num_blocks_calcp = block_size_calcp == 0 ? 0 : (num_components + block_size_calcp - 1) / block_size_calcp;

		std::vector<uint32_t> last_round_possible_progress(num_components, 0);

		std::vector<uint32_t> sim_comb;
		std::vector<uint32_t> comb_type_counts;
		std::vector<uint32_t> comb_type_offsets;
		std::vector<uint32_t> sim_seq;
		std::vector<uint32_t> seq_type_counts;
		std::vector<uint32_t> seq_type_offsets;

		uint32_t* sim_comb_dev;
		uint32_t* comb_type_counts_dev;
		uint32_t* comb_type_offsets_dev;
		uint32_t* sim_seq_dev;
		uint32_t* seq_type_counts_dev;
		uint32_t* seq_type_offsets_dev;

		cu(hipMalloc(&sim_comb_dev, num_components_comb * sizeof(uint32_t)));
		cu(hipMalloc(&comb_type_counts_dev, num_component_types * sizeof(uint32_t)));
		cu(hipMalloc(&comb_type_offsets_dev, num_component_types * sizeof(uint32_t)));
		cu(hipMalloc(&sim_seq_dev, num_components_seq * sizeof(uint32_t)));
		cu(hipMalloc(&seq_type_counts_dev, num_component_types * sizeof(uint32_t)));
		cu(hipMalloc(&seq_type_offsets_dev, num_component_types * sizeof(uint32_t)));

		while (!simulation_finished) { //run simulation rounds until simulation is finished
			int finished_components = 0;

			calc_sim_progress<<<num_blocks_calcp, block_size_calcp>>>(components_dev, num_components); //calculate progress for components
			cu_kernel_errcheck_nosync();
			copy_component_progress_from_device(); //copy component progress to host

			//combinatorial components
			sim_comb.clear();
			comb_type_counts.clear();
			comb_type_offsets.clear();
			comb_type_offsets.push_back(0);
			uint32_t last_type = 0;
			uint32_t comb_sim_words = 0;
			for (uint32_t i = 0; i < num_components_comb; i++) {
				CircuitComponent* comp = components_host[i];
				uint32_t ctype = comp->component_type;

				if (comp->next_sim_progress() == last_round_possible_progress[i]) { //check and mark finished components
					finished_components++;
					continue;
				}
				last_round_possible_progress[i] = comp->next_sim_progress();

				auto words = comp->next_sim_progress_word() - comp->current_sim_progress_word();
				if (words > comb_sim_words) comb_sim_words = words; //remember largest number of words to be simulated

				//arrange components by types and remember counts and offsets
				if (ctype == last_type) {
					comb_type_counts.back()++;
				}
				else {
					if (!comb_type_counts.empty()) comb_type_offsets.push_back(comb_type_offsets.back() + comb_type_counts.back());
					comb_type_counts.push_back(1);
					last_type = ctype;
				}
				sim_comb.push_back(i);
			}

			if (!sim_comb.empty()) { //if combinatorial components need to be simulated
				//copy component pointers, counts, offsets to device
				cu(hipMemcpy(sim_comb_dev, sim_comb.data(), sim_comb.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
				cu(hipMemcpy(comb_type_counts_dev, comb_type_counts.data(), comb_type_counts.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
				cu(hipMemcpy(comb_type_offsets_dev, comb_type_offsets.data(), comb_type_offsets.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

				uint32_t block_size_y = __min(comb_sim_words, 64); //block size y: words to simulate

				auto num_threads_comp = *std::max_element(comb_type_counts.begin(), comb_type_counts.end());
				uint32_t block_size_x = __min(num_threads_comp, 256 / block_size_y); //block size x: components of same type to simulate

				dim3 block_size(block_size_x, block_size_y);

				//grid size x and y: component/word count split into multiple blocks
				//grid size z: component types
				dim3 grid_size((num_threads_comp + block_size_x - 1) / block_size_x, (comb_sim_words + block_size_y - 1) / block_size_y, comb_type_counts.size());

				//simulate
				exec_sim_step<<<grid_size, block_size>>>(components_dev, sim_comb_dev, comb_type_counts_dev, comb_type_offsets_dev);
				cu_kernel_errcheck();

				//mark step as finished
				uint32_t block_size_fin = __min(sim_comb.size(), 256);
				uint32_t num_blocks_fin = (sim_comb.size() + block_size_fin - 1) / block_size_fin;
				finish_sim_step<<<num_blocks_fin, block_size_fin>>>(components_dev, sim_comb_dev, sim_comb.size());
				cu_kernel_errcheck();
			}

			//sequential components, similar to combinatorial as shown above
			sim_seq.clear();
			seq_type_counts.clear();
			seq_type_offsets.clear();
			seq_type_offsets.push_back(0);
			for (uint32_t i = num_components_comb; i < num_components; i++) {
				CircuitComponent* comp = components_host[i];
				uint32_t ctype = comp->component_type;

				if (comp->next_sim_progress() == last_round_possible_progress[i]) { //check and mark finished components
					finished_components++;
					continue;
				}
				last_round_possible_progress[i] = comp->next_sim_progress();

				if (ctype == last_type) {
					seq_type_counts.back()++;
				}
				else {
					if (!seq_type_counts.empty()) seq_type_offsets.push_back(seq_type_offsets.back() + seq_type_counts.back());
					seq_type_counts.push_back(1);
					last_type = ctype;
				}
				sim_seq.push_back(i);
			}

			if (!sim_seq.empty()) {
				cu(hipMemcpy(sim_seq_dev, sim_seq.data(), sim_seq.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
				cu(hipMemcpy(seq_type_counts_dev, seq_type_counts.data(), seq_type_counts.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
				cu(hipMemcpy(seq_type_offsets_dev, seq_type_offsets.data(), seq_type_offsets.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

				auto max_num_threads = *std::max_element(seq_type_counts.begin(), seq_type_counts.end());
				uint32_t block_size = __min(max_num_threads, 256); //only one thread per component -> block size y = 1, grid size y = 1
				dim3 grid_size((max_num_threads + block_size - 1) / block_size, 1, seq_type_counts.size());

				exec_sim_step<<<grid_size, block_size>>>(components_dev, sim_seq_dev, seq_type_counts_dev, seq_type_offsets_dev);
				cu_kernel_errcheck();

				uint32_t block_size_fin = __min(sim_seq.size(), 256);
				uint32_t num_blocks_fin = (sim_seq.size() + block_size_fin - 1) / block_size_fin;
				finish_sim_step<<<num_blocks_fin, block_size_fin>>>(components_dev, sim_seq_dev, sim_seq.size());
				cu_kernel_errcheck();
			}

			if (finished_components == num_components) simulation_finished = true; //done if all components finished
		}

		cu(hipFree(sim_comb_dev));
		cu(hipFree(comb_type_counts_dev));
		cu(hipFree(comb_type_offsets_dev));
		cu(hipFree(sim_seq_dev));
		cu(hipFree(seq_type_counts_dev));
		cu(hipFree(seq_type_offsets_dev));
	}

	StochasticNumber& StochasticCircuit::get_net_value(uint32_t net) {
		if (net >= num_nets) throw std::runtime_error("get_net_value: Invalid net index.");

		return net_numbers[net];
	}

	const StochasticNumber& StochasticCircuit::get_net_value(uint32_t net) const {
		if (net >= num_nets) throw std::runtime_error("get_net_value: Invalid net index.");

		return net_numbers[net];
	}

	double StochasticCircuit::get_net_value_unipolar(uint32_t net) const {
		if (net >= num_nets) throw std::runtime_error("get_net_value_unipolar: Invalid net index.");
		if (net_progress_host[net] == 0) throw std::runtime_error("get_net_value_unipolar: Given net is undefined for the entire simulation time span.");

		return net_numbers[net].get_value_unipolar();
	}

	double StochasticCircuit::get_net_value_bipolar(uint32_t net) const {
		if (net >= num_nets) throw std::runtime_error("get_net_value_bipolar: Invalid net index.");
		if (net_progress_host[net] == 0) throw std::runtime_error("get_net_value_bipolar: Given net is undefined for the entire simulation time span.");

		return net_numbers[net].get_value_bipolar();
	}

	void StochasticCircuit::get_net_values_cuda(uint32_t* nets, double* values_unipolar, uint32_t count, uint32_t length, bool copy) {
		if (host_only) throw std::runtime_error("get_net_values_cuda: Cannot be used in host-only circuits.");
		if (length == 0 || length % 32 != 0) throw std::runtime_error("get_net_values_cuda: Length must be a multiple of 32 and greater than zero.");
		if (length > sim_length) throw std::runtime_error("get_net_values_cuda: Length exceeds the simulation time span.");
		if (count == 0) throw std::runtime_error("get_net_values_cuda: Count must be greater than zero.");

		auto netvalue_ptrs = (uint32_t**)malloc(count * sizeof(uint32_t*)); //calculate device pointers for net data
		for (uint32_t i = 0; i < count; i++) {
			uint32_t net = nets[i];
			if (net >= num_nets) throw std::runtime_error("get_net_values_cuda: Invalid net index.");
			netvalue_ptrs[i] = (uint32_t*)((char*)net_values_dev + (net * net_values_dev_pitch));
		}

		auto word_length = length / 32;

		if (copy) copy_data_to_device(length);

		StochasticNumber::evaluate_bitstreams_cuda(netvalue_ptrs, length, values_unipolar, count);

		free(netvalue_ptrs);
	}

	void StochasticCircuit::get_net_values_cuda(uint32_t* nets, double* values_unipolar, uint32_t count, bool copy) {
		get_net_values_cuda(nets, values_unipolar, count, sim_length, copy);
	}

	void StochasticCircuit::get_net_values_cuda(uint32_t first_net, double* values_unipolar, uint32_t count, uint32_t length, bool copy) {
		if (host_only) throw std::runtime_error("get_net_values_cuda: Cannot be used in host-only circuits.");
		if (length == 0 || length % 32 != 0) throw std::runtime_error("get_net_values_cuda: Length must be a multiple of 32 and greater than zero.");
		if (length > sim_length) throw std::runtime_error("get_net_values_cuda: Length exceeds the simulation time span.");
		if (count == 0) throw std::runtime_error("get_net_values_cuda: Count must be greater than zero.");
		if (first_net + count > num_nets) throw std::runtime_error("get_net_values_cuda: Invalid first net index and/or too many nets.");

		auto netvalue_ptrs = (uint32_t**)malloc(count * sizeof(uint32_t*)); //calculate device pointers for net data
		for (uint32_t i = 0; i < count; i++) {
			netvalue_ptrs[i] = (uint32_t*)((char*)net_values_dev + ((first_net + i) * net_values_dev_pitch));
		}

		auto word_length = length / 32;

		if (copy) copy_data_to_device(length);

		StochasticNumber::evaluate_bitstreams_cuda(netvalue_ptrs, length, values_unipolar, count);

		free(netvalue_ptrs);
	}

	void StochasticCircuit::get_net_values_cuda(uint32_t first_net, double* values_unipolar, uint32_t count, bool copy) {
		get_net_values_cuda(first_net, values_unipolar, count, sim_length, copy);
	}

	CircuitComponent* StochasticCircuit::get_component(uint32_t index) {
		if (index >= num_components) throw std::runtime_error("get_component: Invalid component index.");

		return components_host[components_host_index[index]];
	}

	void StochasticCircuit::copy_component_progress_from_device() {
		cu(hipMemcpy(component_progress_host, component_progress_dev, 2 * num_components * sizeof(uint32_t), hipMemcpyDeviceToHost));
	}

	void StochasticCircuit::set_scheduler(Scheduler* scheduler) {
		if (scheduler != nullptr && scheduler->is_compiled()) throw std::runtime_error("set_scheduler: Already compiled schedulers cannot be assigned to a circuit.");

		scheduler->compile(this);
		delete this->scheduler;
		this->scheduler = scheduler;
	}

	const Scheduler* StochasticCircuit::get_scheduler() {
		return scheduler;
	}

}
