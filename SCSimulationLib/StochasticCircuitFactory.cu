﻿#include "cuda_base.cuh"

#include <algorithm>

#include "StochasticCircuitFactory.cuh"
#include "StochasticCircuit.cuh"
#include "StochasticNumber.cuh"
#include "CircuitComponent.cuh"
#include "CombinatorialComponent.cuh"
#include "SequentialComponent.cuh"

namespace scsim {

	StochasticCircuitFactory::StochasticCircuitFactory(bool host_only) : host_only(host_only) {
		reset();
	}

	StochasticCircuitFactory::~StochasticCircuitFactory() {
		for (size_t i = 0; i < components.size(); i++) {
			delete components[i];
		}

		if (net_numbers != nullptr) {
			for (uint32_t i = 0; i < num_nets; i++) {
				net_numbers[i].~StochasticNumber();
			}
		}

		if (!host_only) {
			cu_ignore_error(hipHostUnregister(net_values_host));
			cu_ignore_error(hipHostUnregister(net_progress_host));
			cu_ignore_error(hipHostUnregister(component_progress_host));
			cu_ignore_error(hipHostUnregister(component_array_host));

			cu_ignore_error(hipFree(net_values_dev));
			cu_ignore_error(hipFree(net_progress_dev));
			cu_ignore_error(hipFree(components_dev));
			cu_ignore_error(hipFree(component_progress_dev));
			cu_ignore_error(hipFree(component_array_dev));
			cu_ignore_error(hipFree(component_io_dev));
			cu_ignore_error(hipFree(component_io_offsets_dev));

			cu_ignore_error(hipHostFree(dev_offset_scratchpad));
			cu_ignore_error(hipHostFree(dev_pointers_scratchpad));
		}

		free(net_values_host);
		free(net_progress_host);
		free(components_host);
		free(component_progress_host);
		free(component_array_host);
		free(component_io_host);
		free(component_io_offsets_host);
		free(net_numbers);
	}

	void StochasticCircuitFactory::reset() {
		sim_length = 0;
		num_nets = 0;
		num_comb_comp = 0;
		num_seq_comp = 0;
		components.clear();
		driven_nets.clear();
		max_component_size = 0;
		max_component_align = 0;
		component_io.clear();

		net_values_host = nullptr;
		net_values_dev = nullptr;
		net_progress_host = nullptr;
		net_progress_dev = nullptr;
		components_host = nullptr;
		components_dev = nullptr;
		component_progress_host = nullptr;
		component_progress_dev = nullptr;
		component_array_host = nullptr;
		component_array_dev = nullptr;
		component_io_host = nullptr;
		component_io_dev = nullptr;
		component_io_offsets_host = nullptr;
		component_io_offsets_dev = nullptr;
		dev_offset_scratchpad = nullptr;
		dev_pointers_scratchpad = nullptr;
		net_numbers = nullptr;
	}

	StochasticCircuit* StochasticCircuitFactory::create_circuit() {
		if (sim_length == 0) throw std::exception("create_circuit: Simulation time span has not been set.");
		if (num_nets == 0 || components.size() == 0) throw std::exception("create_circuit: Factory is empty, so circuit creation is impossible.");

		StochasticCircuit* circuit = nullptr;

		size_t sim_length_words = (sim_length + 31) / 32;

		//host-side circuit state
		net_values_host = (uint32_t*)malloc(sim_length_words * num_nets * sizeof(uint32_t));
		net_progress_host = (uint32_t*)calloc(num_nets, sizeof(uint32_t));
		components_host = (CircuitComponent**)malloc(components.size() * sizeof(CircuitComponent*));
		components_dev = nullptr;
		component_progress_host = (uint32_t*)calloc(2 * components.size(), sizeof(uint32_t));
		component_progress_dev = nullptr;
		component_io_host = (uint32_t*)malloc(component_io.size() * sizeof(uint32_t));
		component_io_dev = nullptr;
		component_io_offsets_host = (size_t*)malloc(component_io.size() * sizeof(size_t));
		component_io_offsets_dev = nullptr;
		net_numbers = (StochasticNumber*)calloc(num_nets, sizeof(StochasticNumber));

		if (net_values_host == nullptr || net_progress_host == nullptr || components_host == nullptr || component_progress_host == nullptr ||
			component_io_host == nullptr || component_io_offsets_host == nullptr || net_numbers == nullptr) throw std::exception("create_circuit: Host-side state allocation failed.");

		size_t component_pitch;
		size_t component_array_dev_pitch;

		std::stable_sort(components.begin(), components.end(), [](auto a, auto b) { return a->component_type < b->component_type; }); //sort components by type for automatic grouping during simulation
		memcpy(components_host, components.data(), components.size() * sizeof(CircuitComponent*));
		memcpy(component_io_host, component_io.data(), component_io.size() * sizeof(uint32_t));

		uint32_t num_component_types = 0;
		uint32_t last_type = 0;
		for (uint32_t i = 0; i < components.size(); i++) { //count component types
			uint32_t type = components[i]->component_type;
			if (type != last_type) {
				num_component_types++;
				last_type = type;
			}
		}

		for (uint32_t i = 0; i < num_nets; i++) { //create stochastic numbers corresponding to nets
			new (net_numbers + i) StochasticNumber(net_values_host + (sim_length_words * i), net_progress_host + i, sim_length);
		}

		if (host_only) { //host-only circuit: create circuit and initialize components, that's all
			circuit = new StochasticCircuit(sim_length, num_nets, net_values_host, net_progress_host, num_comb_comp, num_seq_comp, components_host, component_progress_host, num_component_types,
				component_io_host, component_io_offsets_host, net_numbers);
			for (uint32_t i = 0; i < components.size(); i++) {
				components_host[i]->init_with_circuit(circuit, component_progress_host + (2ull * i), nullptr, nullptr);
			}
		}
		else { //device-assisted circuit
			size_t net_pitch_dev;

			//page-lock host-side circuit state if possible
			cu_ignore_error(hipHostRegister(net_values_host, sim_length_words * num_nets * sizeof(uint32_t), hipHostRegisterDefault));
			cu_ignore_error(hipHostRegister(net_progress_host, num_nets * sizeof(uint32_t), hipHostRegisterDefault));
			cu_ignore_error(hipHostRegister(component_progress_host, 2 * components.size() * sizeof(uint32_t), hipHostRegisterDefault));

			//device-side circuit state
			cu(hipMallocPitch(&net_values_dev, &net_pitch_dev, sim_length_words * sizeof(uint32_t), num_nets));
			cu(hipMalloc(&net_progress_dev, num_nets * sizeof(uint32_t)));
			cu(hipMalloc(&components_dev, components.size() * sizeof(CircuitComponent*)));
			cu(hipMalloc(&component_progress_dev, 2 * components.size() * sizeof(uint32_t)));
			cu(hipMalloc(&component_io_dev, component_io.size() * sizeof(uint32_t)));
			cu(hipMalloc(&component_io_offsets_dev, component_io.size() * sizeof(size_t)));

			auto max_comp_size_misalignment = max_component_size % max_component_align;
			if (max_comp_size_misalignment == 0) component_pitch = max_component_size;
			else component_pitch = max_component_size - max_comp_size_misalignment + max_component_align;

			//component array, host side, page-locked if possible
			component_array_host = (char*)malloc(components.size() * component_pitch);
			if (component_array_host == nullptr) throw std::exception("create_circuit: Host-side state allocation failed.");
			cu_ignore_error(hipHostRegister(component_array_host, components.size() * component_pitch, hipHostRegisterDefault));

			//component array, device side
			cu(hipMallocPitch(&component_array_dev, &component_array_dev_pitch, component_pitch, components.size()));

			circuit = new StochasticCircuit(sim_length, num_nets, net_values_host, net_values_dev, net_pitch_dev, net_progress_host, net_progress_dev, num_comb_comp, num_seq_comp, components_host,
				components_dev, component_array_host, component_pitch, component_array_dev, component_array_dev_pitch, component_progress_host, component_progress_dev, num_component_types,
				component_io_host, component_io_dev, component_io_offsets_host, component_io_offsets_dev, net_numbers);

			cu(hipHostMalloc(&dev_offset_scratchpad, component_io.size() * sizeof(size_t)));
			cu(hipHostMalloc(&dev_pointers_scratchpad, components.size() * sizeof(CircuitComponent*)));

			for (uint32_t i = 0; i < components.size(); i++) {
				auto comp = components_host[i];

				components_host[i] = (CircuitComponent*)(component_array_host + i * component_pitch); //redirect component pointers to component array

				comp->dev_ptr = (CircuitComponent*)(component_array_dev + i * component_array_dev_pitch); //link components's own device pointer to component array
				dev_pointers_scratchpad[i] = comp->dev_ptr; //direct device-side pointers to component array

				comp->init_with_circuit(circuit, component_progress_host + (2ull * i), component_progress_dev + (2ull * i), dev_offset_scratchpad);
				
				memcpy(components_host[i], comp, comp->mem_obj_size); //copy component to component array
			}

			cu(hipMemcpy(component_io_dev, component_io_host, component_io.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
			cu(hipMemcpy(component_io_offsets_dev, dev_offset_scratchpad, component_io.size() * sizeof(size_t), hipMemcpyHostToDevice));
			cu(hipMemcpy(components_dev, dev_pointers_scratchpad, components.size() * sizeof(CircuitComponent*), hipMemcpyHostToDevice));

			cu_ignore_error(hipHostFree(dev_offset_scratchpad));
			cu_ignore_error(hipHostFree(dev_pointers_scratchpad));
			dev_offset_scratchpad = nullptr;
			dev_pointers_scratchpad = nullptr;

			for (auto comp : components) operator delete(comp); //free original allocation for components (without destroying the actual components)

			circuit->reset_circuit();
		}

		reset();

		return circuit;
	}

	void StochasticCircuitFactory::set_sim_length(uint32_t sim_length) {
		if (sim_length == 0) throw std::exception("set_sim_length: Simulation time span must be greater than zero.");
		this->sim_length = sim_length;
	}

	uint32_t StochasticCircuitFactory::add_net() {
		driven_nets.push_back(false); //new net initially undriven
		return num_nets++;
	}

	std::pair<uint32_t, uint32_t> StochasticCircuitFactory::add_nets(uint32_t count) {
		if (count == 0) throw std::exception("add_nets: Number of added nets must be greater than zero.");
		driven_nets.resize(driven_nets.size() + count, false); //new nets initially undriven
		auto first = num_nets;
		num_nets += count;
		return std::make_pair(first, num_nets - 1);
	}

	uint32_t StochasticCircuitFactory::add_component(CombinatorialComponent* component) {
		uint32_t index = add_component_internal(component);
		num_comb_comp++;
		return index;
	}

	uint32_t StochasticCircuitFactory::add_component_internal(CircuitComponent* component) {
		for (size_t i = 0; i < component->num_inputs; i++) {
			//disallow invalid/nonexistent nets
			if (component->inputs_host[i] >= num_nets) throw std::exception("add_component: Component references a net that does not exist.");
		}

		for (size_t i = 0; i < component->num_outputs; i++) {
			auto net = component->outputs_host[i];
			//disallow invalid/nonexistent nets and multiple outputs per net
			if (net >= num_nets) throw std::exception("add_component: Component references a net that does not exist.");
			if (driven_nets[net]) throw std::exception("add_component: An output net is already used as another component's output.");
		}

		for (size_t i = 0; i < component->num_outputs; i++) {
			auto net = component->outputs_host[i];
			driven_nets[net] = true; //mark all output nets as driven
		}

		//remember largest component memory size and alignment in circuit (minimum component array pitch)
		if (component->mem_obj_size > max_component_size) max_component_size = component->mem_obj_size;
		if (component->mem_align > max_component_align) max_component_align = component->mem_align;

		components.push_back(component);

		return components.size() - 1;
	}

}
