﻿#include "cuda_base.cuh"

#include <iostream>
#include <fstream>
#include "StanhTestbench.cuh"
#include "SquarerTestbench.cuh"
#include "InverterTestbench.cuh"
#include "MuxNTestbench.cuh"

constexpr uint32_t MIN_SN_LENGTH = 1 << 11;

void runBench(Testbench* bench, const char* filename) {
	auto csv = bench->run();

	std::ofstream file;
	file.open(filename);
	file << csv;
	file.close();

	delete bench;
}

void run() {
	cu(hipSetDevice(0));

	runBench(new MuxNTestbench(MIN_SN_LENGTH, 13, 10), "results\\muxn.csv");
	runBench(new InverterTestbench(MIN_SN_LENGTH, 14, 10), "results\\inverter.csv");
	runBench(new SquarerTestbench(MIN_SN_LENGTH, 14, 10), "results\\squarer.csv");
	runBench(new StanhTestbench(MIN_SN_LENGTH, 6, 14, 10), "results\\stanh.csv");
}

int main() {
	try {
		run();
	} catch (std::exception e) {
		std::cerr << e.what();
	}

	return 0;
}